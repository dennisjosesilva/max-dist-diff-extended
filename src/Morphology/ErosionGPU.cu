#include "hip/hip_runtime.h"
#include <Morphology/ErosionGPU.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>


static void HandleError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))


__device__ int OUT_OF_DOMAIN_VALUE = -1;

__global__ void erode(
    morphotree::uint32 width,
    morphotree::uint32 height,
    morphotree::uint8 *f,
    int *dx, int *dy, 
    unsigned int se_size,    
    int *f_eroded)
{
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;  


  if (0 <= px && px < width && 0 <= py && py < height) {      
    // If thread produces a valid pixel
    morphotree::uint32 pidx = width*py + px;
    int minValue = static_cast<int>(f[pidx]);

    for (int i = 0; i < se_size; i++) {
      int qx = px + dx[i];
      int qy = py + dy[i];
      
      if (0 <= qx && qx < width && 0 <= qy && qy < height) {
        morphotree::uint32 qidx = qy * width + qx;  
        if (static_cast<int>(f[qidx]) < minValue) {
          minValue = static_cast<int>(f[qidx]);
        }
      }
      else {
        minValue = OUT_OF_DOMAIN_VALUE;
        break;
      }    
    }
    
    f_eroded[pidx] = minValue;
  }  
}

StructuringElement::StructuringElement(const std::vector<int> &pdx,
  const std::vector<int> &pdy)
  : dx{pdx}, dy{pdy}
{}

StructuringElement::StructuringElement(std::vector<int> &&pdx, 
  std::vector<int> &&pdy)
  : dx{pdx}, dy{pdy}
{}

StructuringElement StructuringElement::UnitCross()
{
  return StructuringElement{
    {-1,  0,  1,  0}, // dx
    { 0, -1,  0,  1}  // dy
  };
}

StructuringElement StructuringElement::UnitBox()
{
  return StructuringElement{
    {-1, -1,  0,  1,  1,  1,  0, -1}, // dx
    { 0, -1, -1, -1,  0,  1,  1,  1}  // dy
  };
}


ErosionGPU::ErosionGPU(const Box& domain, const std::vector<uint8> &f)
  : domain_{domain}, f_{f}
{}

std::vector<int> ErosionGPU::compute(const StructuringElement &se) const
{  
  uint8 *f_dev;
  int *f_eroded_dev;
  int *dx_dev,  *dy_dev;
  
  // Allocate required memory in GPU
  HANDLE_ERROR(hipMalloc((void**)&f_dev, domain_.numberOfPoints() * sizeof(uint8)));
  HANDLE_ERROR(hipMalloc((void**)&f_eroded_dev, static_cast<size_t>(domain_.numberOfPoints()) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dx_dev, se.dx.size() * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dy_dev, se.dy.size() * sizeof(int)));

  // copy CPU to GPU
  HANDLE_ERROR(hipMemcpy(f_dev, f_.data(), static_cast<size_t>(domain_.numberOfPoints()) * sizeof(uint8), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dx_dev, se.dx.data(), se.dx.size() * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dy_dev, se.dy.data(), se.dy.size() * sizeof(int), hipMemcpyHostToDevice));

  dim3 threadDim{16, 16, 1};
  // dim3 blockDim{64, 64, 1};
  dim3 blockDim{ (15 + domain_.width()) / (threadDim.x), 
     (15 + domain_.height()) / (threadDim.y) };


  std::cout << "blockDim: " << blockDim.x << ", " << blockDim.y << std::endl;

  // Compute Block size and thread size
  erode<<<blockDim, threadDim>>>(
    domain_.width(), domain_.height(),  // domain
    f_dev,                              // f
    dx_dev,          dy_dev,            // dx, dy
    se.dx.size(),                       // se width
    f_eroded_dev);                      // output

  
  // copy back data from GPU to CPU  
  std::vector<int> f_eroded(domain_.numberOfPoints());
  HANDLE_ERROR(hipMemcpy(f_eroded.data(), f_eroded_dev, 
    domain_.numberOfPoints() * sizeof(int), hipMemcpyDeviceToHost));
  
  return f_eroded;
}

std::vector<int> erodeGPU(const morphotree::Box &domain,
  const std::vector<morphotree::uint8> &f,
  const StructuringElement &se)
{
  ErosionGPU erosion{domain, f};
  return erosion.compute(se);
}